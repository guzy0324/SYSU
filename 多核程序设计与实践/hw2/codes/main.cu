#include "hip/hip_runtime.h"
# include "entropy.h"
# include <iostream>
# include <windows.h>
using namespace std;

# define M (2048)
# define N (2048)
# define SEED (0)
# define EPOCHS (100)

int in_host[M * N];
float out_host[M * N];

int main()
{
    init_data(in_host, M, N, SEED);
    int** in, * in_data;
    float** out, * out_data;
    hipMalloc((void**)&in_data, sizeof(in_host));
    hipMalloc((void**)&in, sizeof(int*) * M);
    hipMalloc((void**)&out_data, sizeof(out_host));
    hipMalloc((void**)&out, sizeof(float*) * M);
    init_matrix<int><<<divup(M, 64), 64>>>(in, in_data, M, N);
    init_matrix<float><<<divup(M, 64), 64>>>(out, out_data, M, N);
    hipMemcpy(in_data, in_host, sizeof(in_host), hipMemcpyHostToDevice);
    dim3 block(BDIM_X, BDIM_Y, 1);
    dim3 grid(divup(N, BDIM_X), divup(M, BDIM_Y), 1);
	LARGE_INTEGER t1, t2, tc;
	QueryPerformanceFrequency(&tc);

    cout << "M x N: " << M << " x " << N << endl;
    cout << "BlockDim = " << BDIM_Y << " x " << BDIM_X << endl << endl;

    QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
        entropy <<< grid, block >>> (out, in, M, N);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    hipMemcpy(out_host, out_data, sizeof(out_host), hipMemcpyDeviceToHost);
    cout << "Baseline:" << endl;
    cout << "Correctness: " << check_result(out_host, in_host, M, N) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;

    bind_texture(in_data, M, N);
    QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
        entropy_texture <<< grid, block >>> (out, M, N);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    hipMemcpy(out_host, out_data, sizeof(out_host), hipMemcpyDeviceToHost);
    hipUnbindTexture(tex);
    cout << "Texture memory:" << endl;
    cout << "Correctness: " << check_result(out_host, in_host, M, N) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;

    QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
        entropy_shared <<< grid, block >>> (out, in, M, N);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    hipMemcpy(out_host, out_data, sizeof(out_host), hipMemcpyDeviceToHost);
    cout << "Shared memory:" << endl;
    cout << "Correctness: " << check_result(out_host, in_host, M, N) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;

    QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
        entropy_shared_log <<< grid, block >>> (out, in, M, N);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    hipMemcpy(out_host, out_data, sizeof(out_host), hipMemcpyDeviceToHost);
    cout << "Shared memory with log table:" << endl;
    cout << "Correctness: " << check_result(out_host, in_host, M, N) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;

    QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
        entropy_shared_log_ldg <<< grid, block >>> (out, in, M, N);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    hipMemcpy(out_host, out_data, sizeof(out_host), hipMemcpyDeviceToHost);
    cout << "Shared memory with log table accessed through read-only memory:" << endl;
    cout << "Correctness: " << check_result(out_host, in_host, M, N) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;

    QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
        entropy_shared_log_shared <<< grid, block >>> (out, in, M, N);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    hipMemcpy(out_host, out_data, sizeof(out_host), hipMemcpyDeviceToHost);
    cout << "Shared memory with log table accessed through shared memory:" << endl;
    cout << "Correctness: " << check_result(out_host, in_host, M, N) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;
}

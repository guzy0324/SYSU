#include "../hw1.h"
#include <iostream>
#include <windows.h>
using namespace std;

#define M 1024
#define N 1024
#define BS1 32
#define BS2 32
#define W 8
#define H 8
#define EPOCHS 128

int main()
{
    dim3 block_2d(W, H, 1);
    dim3 grid_2d(divup(M, W), divup(N, H), 1);
    dim3 block_1d(W * H, 1);
    dim3 grid_1d(divup(M * N, W * H), 1, 1);

	LARGE_INTEGER t1, t2, tc;
	QueryPerformanceFrequency(&tc);

    int* A_data, ** A, *B_data, ** B, * C_data, ** C;
    hipMalloc((void**)&A_data, sizeof(int) * N * M);
    hipMalloc((void**)&B_data, sizeof(int) * N * M);
    hipMalloc((void**)&C_data, sizeof(int) * N * M);
    hipMalloc((void**)&A, sizeof(int*) * N);
    hipMalloc((void**)&B, sizeof(int*) * N);
    hipMalloc((void**)&C, sizeof(int*) * N);
    init_matrix <<< divup(N, BS2), BS2 >>> (A, A_data, N, M);
    init_matrix <<< divup(N, BS2), BS2 >>> (B, B_data, N, M);
    init_matrix <<< divup(N, BS2), BS2 >>> (C, C_data, N, M);

    init_data <<< divup(N * M, BS1), BS1 >>> (A_data, N, M);
    init_data <<< divup(N * M, BS1), BS1 >>> (B_data, N, M);
    init_data <<< divup(N * M, BS1), BS1 >>> (C_data, N, M);
    hipDeviceSynchronize();
	QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
		matrix_add_1 <<< grid_2d, block_2d >>> (A, B, C, N, M);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    cout << "2D array with 2D grid and block:" << endl;
    cout << "Correctness: " << check_matrix(A_data, B_data, C_data, N, M) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;

    init_data <<< divup(N * M, BS1), BS1 >>> (A_data, N, M);
    init_data <<< divup(N * M, BS1), BS1 >>> (B_data, N, M);
    init_data <<< divup(N * M, BS1), BS1 >>> (C_data, N, M);
    hipDeviceSynchronize();
	QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
        matrix_add_2 <<< grid_2d, block_2d >>> (A_data, B_data, C_data, N, M);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    cout << "1D array with 2D grid and block:" << endl;
    cout << "Correctness: " << check_matrix(A_data, B_data, C_data, N, M) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;

    init_data <<< divup(N * M, BS1), BS1 >>> (A_data, N, M);
    init_data <<< divup(N * M, BS1), BS1 >>> (B_data, N, M);
    init_data <<< divup(N * M, BS1), BS1 >>> (C_data, N, M);
    hipDeviceSynchronize();
	QueryPerformanceCounter(&t1);
    for (int i = 0; i < EPOCHS; ++i)
    {
        matrix_add_3 <<< grid_1d, block_1d >>> (A_data, B_data, C_data, N, M);
        hipDeviceSynchronize();
    }
	QueryPerformanceCounter(&t2);
    cout << "1D array with 1D grid and block:" << endl;
    cout << "Correctness: " << check_matrix(A_data, B_data, C_data, N, M) << endl;
    cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
    cout << endl;
}

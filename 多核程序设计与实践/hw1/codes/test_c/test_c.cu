#include "../hw1.h"
#include <iostream>
#include <windows.h>
using namespace std;

#define M 1024
#define N 1024
#define BS1 32
#define BS2 32
#define BS 64
#define EPOCHS 128

int main()
{
	LARGE_INTEGER t1, t2, tc;
	QueryPerformanceFrequency(&tc);

    int* A_data, ** A, *B_data, ** B, * C_data, ** C;
    hipMallocManaged((void**)&A_data, sizeof(int) * N * M);
    hipMallocManaged((void**)&B_data, sizeof(int) * N * M);
    hipMallocManaged((void**)&C_data, sizeof(int) * N * M);
    hipMallocManaged((void**)&A, sizeof(int*) * N);
    hipMallocManaged((void**)&B, sizeof(int*) * N);
    hipMallocManaged((void**)&C, sizeof(int*) * N);
    init_matrix <<< divup(N, BS2), BS2 >>> (A, A_data, N, M);
    init_matrix <<< divup(N, BS2), BS2 >>> (B, B_data, N, M);
    init_matrix <<< divup(N, BS2), BS2 >>> (C, C_data, N, M);

    for (int k = 1; k <= 4; k += 1)
    {
        init_data <<< divup(N * M, BS1), BS1 >>> (A_data, N, M);
        init_data <<< divup(N * M, BS1), BS1 >>> (B_data, N, M);
        init_data <<< divup(N * M, BS1), BS1 >>> (C_data, N, M);
        hipDeviceSynchronize();
        QueryPerformanceCounter(&t1);
        for (int i = 0; i < EPOCHS; ++i)
        {
            matrix_add_4 <<< divup(divup(M * N, k), BS), BS >>> (A_data, B_data, C_data, N, M, k);
            hipDeviceSynchronize();
        }
        QueryPerformanceCounter(&t2);
        cout << k << " elements:" << endl;
        cout << "Correctness: " << check_matrix(A_data, B_data, C_data, N, M) << endl;
        cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
        cout << endl;
    }
}

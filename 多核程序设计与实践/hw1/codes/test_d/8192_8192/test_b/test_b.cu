#include "../../../hw1.h"
#include <iostream>
#include <windows.h>
using namespace std;

#define M 8192
#define N 8192
#define BS1 32
#define BS2 32
#define EPOCHS 128

int BS[] = {16, 32, 63, 64, 65, 128};

int main()
{
	LARGE_INTEGER t1, t2, tc;
	QueryPerformanceFrequency(&tc);

    int* A_data, ** A, *B_data, ** B, * C_data, ** C;
    hipMallocManaged((void**)&A_data, sizeof(int) * N * M);
    hipMallocManaged((void**)&B_data, sizeof(int) * N * M);
    hipMallocManaged((void**)&C_data, sizeof(int) * N * M);
    hipMallocManaged((void**)&A, sizeof(int*) * N);
    hipMallocManaged((void**)&B, sizeof(int*) * N);
    hipMallocManaged((void**)&C, sizeof(int*) * N);
    init_matrix <<< divup(N, BS2), BS2 >>> (A, A_data, N, M);
    init_matrix <<< divup(N, BS2), BS2 >>> (B, B_data, N, M);
    init_matrix <<< divup(N, BS2), BS2 >>> (C, C_data, N, M);

    for (int BS_i = 0; BS_i < sizeof(BS) / sizeof(int); ++BS_i)
    {
        init_data <<< divup(N * M, BS1), BS1 >>> (A_data, N, M);
        init_data <<< divup(N * M, BS1), BS1 >>> (B_data, N, M);
        init_data <<< divup(N * M, BS1), BS1 >>> (C_data, N, M);
        hipDeviceSynchronize();
        QueryPerformanceCounter(&t1);
        for (int i = 0; i < EPOCHS; ++i)
        {
            matrix_add_3 <<< divup(M * N, BS[BS_i]), BS[BS_i] >>> (A_data, B_data, C_data, N, M);
            hipDeviceSynchronize();
        }
        QueryPerformanceCounter(&t2);
        cout << "Block size: " << BS[BS_i] << endl;
        cout << "Correctness: " << check_matrix(A_data, B_data, C_data, N, M) << endl;
        cout << "Efficiency: " << (t2.QuadPart - t1.QuadPart) * 1.0 / EPOCHS / tc.QuadPart << " us" << endl;
        cout << endl;
    }
}

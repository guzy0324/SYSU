#include "hip/hip_runtime.h"
#include "error_checks.h" // Macros CUDA_CHECK and CHECK_ERROR_MSG
#include <cmath>
#include <cstdio>
#include <dos.h>

__global__ void vector_add(double *C, const double *A, const double *B, int N)
{
    // Add the kernel code
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    /*if (idx == N - 1)
    {
		for (int i = 0; i < 1000; i++);
    }*/
    // Do not try to access past the allocated memory
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}

int main(void)
{
    const int N = 20;
    const int ThreadsInBlock = 128;
    //const int ThreadsInBlock = 1024;
    //const int ThreadsInBlock = 1025;
    double *dA, *dB, *dC;
    double hA[N], hB[N], hC[N];

    for (int i = 0; i < N; ++i)
    {
        hA[i] = (double)i;
        hB[i] = (double)i * i;
    }

    /* 
    Add memory allocations and copies. Wrap your runtime function
    calls with CUDA_CHECK( ) macro
    */
    CUDA_CHECK(hipMalloc((void **)&dA, sizeof(double) * N));
    CUDA_CHECK(hipMalloc((void **)&dB, sizeof(double) * N));
    CUDA_CHECK(hipMalloc((void **)&dC, sizeof(double) * N));
    CUDA_CHECK(hipMemcpy((void *)dA, (void *)hA, N * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy((void *)dB, (void *)hB, N * sizeof(double), hipMemcpyHostToDevice));
    /*hipMalloc((void **)&dA, sizeof(double) * N);
    hipMalloc((void **)&dB, sizeof(double) * N);
    hipMalloc((void **)&dC, sizeof(double) * N);
    hipMemcpy((void *)dA, (void *)hA, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void *)dB, (void *)hB, N * sizeof(double), hipMemcpyHostToDevice);*/
    //printf("%f", *dA);

    // Note the maximum size of threads in a block
    dim3 grid, threads;

    //// Add the kernel call here
    vector_add<<<N / ThreadsInBlock + 1, ThreadsInBlock>>>(dC, dA, dB, N);
    //vector_add<<<N / ThreadsInBlock + 1, ThreadsInBlock>>>(hC, hA, hB, N);

    // Here we add an explicit synchronization so that we catch errors
    // as early as possible. Don't do this in production code!
    hipDeviceSynchronize();
    CHECK_ERROR_MSG("vector_add kernel");

    //// Copy back the results and free the device memory
    CUDA_CHECK(hipMemcpy((void *)hC, (void *)dC, N * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree((void *)dA));
    CUDA_CHECK(hipFree((void *)dB));
    CUDA_CHECK(hipFree((void *)dC));
    /*hipMemcpy((void *)hC, (void *)dC, N * sizeof(double), hipMemcpyDeviceToHost);
    hipFree((void *)dA);
    hipFree((void *)dB);
    hipFree((void *)dC);*/

    for (int i = 0; i < N; i++)
        printf("%5.1f\n", hC[i]);

    return 0;
}